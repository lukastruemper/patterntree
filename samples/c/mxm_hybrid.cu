
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define N 8192
#define K 256
#define M 4096

__global__
void mxv(double* a, double* b, double* c) {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int row = threadIdx.x + blockId * (blockDim.x * blockDim.y);    

    for (int i = 0; i < M; i++)
    {
        c[row * M + i] = 0.0;
        for (int j = 0; j < N; j++) {
            c[row * M + i] += a[row * N + j] * b[i * N + j];
        }
    }
}

int main(int argc, char** argv)
{
    // 1. Memory allocation
    double* a_h = (double *) malloc(N * N * sizeof(double));
    double* b_h = (double *) malloc(M * N * sizeof(double));
    double* c_h = (double *) malloc(M * N * sizeof(double));
    
    double* a_d;
    double* b_d;
    double* c_d;
    hipMalloc(&a_d, (N - K) * N * sizeof(double));
    hipMalloc(&b_d, M * N * sizeof(double));
    hipMalloc(&c_d, M * (N - K) * sizeof(double));


    printf("Allocation done\n");

    // 2. Random initialization
    int seed = 42;
    srand(seed);
    for (int i = 0; i < N * N; i++) {
        a_h[i] = 2.0;
        //m_h[i] = ((double) rand() / (double) RAND_MAX) * 2.0 - 1.0;
    }
    for (int i = 0; i < N * M; i++) {
        b_h[i] = 4.0;
    }

    printf("Random init done\n");

    printf("Starting clock\n");
    // 3. Start clock
    double time = omp_get_wtime();
    
    hipMemcpy(a_d, a_h, (N - K) * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, M * N * sizeof(double), hipMemcpyHostToDevice);

    mxv<<<(N - K) / 256, 256>>>(a_d, b_d, c_d);

    #pragma omp parallel for
    for (int i = N - K; i < N; i++) {
        for (int k = 0; k < M; k++) {
            c_h[i * M + k] = 0.0;
            for (int j = 0; j < N; j++) {
                c_h[i * M + k] += a_h[i * N + j] * b_h[k * N + j];
            }
        }
    }

    hipMemcpy(c_h, c_d, M * (N - K) * sizeof(double), hipMemcpyDeviceToHost);

    // 4. Stop time
    time = omp_get_wtime() - time;
    printf("Stopping clock\n");
    
    // 5. Computation for non-trivialization of code
    for (int i = 0; i < M * N; i++) {
        if (c_h[i] != N * 8.0) {
            printf("ERRROR: %f!", c_h[i]);
        }
    }

    printf("Computation took: %f\n", time);
      
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);
    
    return 0;
}
